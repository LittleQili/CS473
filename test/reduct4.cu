#define FOURMB 2 * 1024 * 1024 - 100
#define BYTES FOURMB * sizeof(int)
#define NTHREADS 128


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__device__ void wrapReduce(volatile int* sdata,int tid){
	sdata[tid] += sdata[tid+32];
	sdata[tid] += sdata[tid+16];
	sdata[tid] += sdata[tid+8];
	sdata[tid] += sdata[tid+4];
	sdata[tid] += sdata[tid+2];
	sdata[tid] += sdata[tid+1];
}

__global__ void reduce(int *g_idata, int *g_odata, unsigned int n)
{
	extern __shared__ int sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	sdata[tid] = (i<n)?g_idata[i]:0;
	sdata[tid] += (i+blockDim.x<n)?g_idata[i + blockDim.x]:0;

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if(tid < 32)wrapReduce(sdata,tid);
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

__host__ int reduce_start(int *h_idata)
{
	int *d_idata;
	int *d_odata;
	int *d_intermediateSums;
	int res = 0;

	hipError_t cudaStatus;

	int nBlocks = FOURMB / NTHREADS / 2;
	cudaStatus = hipMalloc((void **)&d_idata, BYTES);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	cudaStatus = hipMalloc((void **)&d_odata, nBlocks * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	cudaStatus = hipMalloc((void **)&d_intermediateSums, sizeof(int) * nBlocks);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}

	cudaStatus = hipMemcpy(d_idata, h_idata, BYTES, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		exit(1);
	}
	// 为device分配空间及把输入拷入device的global_memory
	struct timespec time_start = {0, 0}, time_end = {0, 0};
	clock_gettime(CLOCK_REALTIME, &time_start);
	for (int idx = 0; idx < 100; idx++)
	{
		dim3 dimBlock(NTHREADS, 1, 1);
		dim3 dimGrid(nBlocks, 1, 1);
		int smemSize = NTHREADS * sizeof(int);
		reduce<<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, FOURMB);
		int s = nBlocks;
		while (s > 1)
		{
			dim3 dimGrid((s + NTHREADS - 1) / NTHREADS, 1, 1);
			cudaStatus = hipMemcpy(d_intermediateSums, d_odata, s * sizeof(int), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy failed!");
				exit(1);
			}
			reduce<<<dimGrid, dimBlock, smemSize>>>(d_intermediateSums, d_odata, s);
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				exit(1);
			}
			s /= (NTHREADS * 2);
		}
		cudaStatus = hipMemcpy(&res, d_odata, sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			exit(1);
		}
	}
	clock_gettime(CLOCK_REALTIME, &time_end);
	double costTime = (time_end.tv_sec - time_start.tv_sec) * 1000 * 1000 * 1000 + time_end.tv_nsec - time_start.tv_nsec;
	printf("duration:%.7lfdms\n", costTime / 1000 / 1000);
	hipFree(d_intermediateSums);
	return res;
}

__host__ int main()
{
	int *h_idata;
	h_idata = (int *)malloc(BYTES);
	for (int i = 0; i < FOURMB; i++)
	{
		h_idata[i] = rand() & 0xff;
	}
	int cpu_result = 0;
	for (int i = 0; i < FOURMB; i++)
	{
		cpu_result += h_idata[i];
	}
	int gpu_result = reduce_start(h_idata);
	printf("cpu_result: %d\n", cpu_result);
	printf("gpu_result: %d\n", gpu_result);
}