#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void print()
{
    unsigned int tid = threadIdx.x;
    unsigned int globalTid = blockDim.x * blockIdx.x + threadIdx.x;
    if (globalTid < 128)
        printf("Hello from %d %d\n", tid, globalTid);
}

int main()
{
    hipError_t cudaStatus;
    print<<<1, 1025>>>();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "printKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        exit(0);
    }
    hipDeviceSynchronize();
}
