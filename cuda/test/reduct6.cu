#define FOURMB 2 * 1024 * 1024
#define BYTES FOURMB * sizeof(int)
#define NTHREADS 128
#define NGRIDS 256


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

template <unsigned int blockSize>
__device__ void wrapReduce(volatile int *sdata, int tid)
{
	if (blockSize >= 64)
		sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32)
		sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16)
		sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8)
		sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4)
		sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2)
		sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce(int *g_idata, int *g_odata, unsigned int n)
{
	extern __shared__ int sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < n)
	{
		sdata[tid] += g_idata[i];
		if (i + blockSize < n) sdata[tid] += g_idata[i + blockSize];
		i += gridSize;
	}

	__syncthreads();

	if (blockSize >= 512 && (tid < 256))
	{
		sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if (blockSize >= 256 && (tid < 128))
	{
		sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if (blockSize >= 128 && (tid < 64))
	{
		sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid < 32)
		wrapReduce<blockSize>(sdata, tid);
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

__host__ int reduce_start(int *h_idata)
{
	int *d_idata;
	int *d_odata;
	int *d_intermediateSums;
	int res = 0;

	hipError_t cudaStatus;

	int nBlocks = FOURMB / NTHREADS / 2;
	cudaStatus = hipMalloc((void **)&d_idata, BYTES);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	cudaStatus = hipMalloc((void **)&d_odata, nBlocks * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}
	cudaStatus = hipMalloc((void **)&d_intermediateSums, sizeof(int) * nBlocks);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}

	cudaStatus = hipMemcpy(d_idata, h_idata, BYTES, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy h_idata failed!");
		exit(1);
	}
	// 为device分配空间及把输入拷入device的global_memory
	struct timespec time_start = {0, 0}, time_end = {0, 0};
	clock_gettime(CLOCK_REALTIME, &time_start);
	for (int idx = 0; idx < 100; idx++)
	{
		dim3 dimBlock(NTHREADS, 1, 1);
		dim3 dimGrid(NGRIDS,1,1);
		// dim3 dimGrid(nBlocks, 1, 1);
		int smemSize = NTHREADS * sizeof(int);
		reduce<NTHREADS><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, FOURMB);
		int s = nBlocks;
		while (s > 1)
		{
			// dim3 dimGrid((s + NTHREADS - 1) / NTHREADS, 1, 1);
			dim3 dimGrid(NGRIDS,1,1);
			cudaStatus = hipMemcpy(d_intermediateSums, d_odata, s * sizeof(int), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "hipMemcpy outTointermediate failed!");
				exit(1);
			}
			reduce<NTHREADS><<<dimGrid, dimBlock, smemSize>>>(d_intermediateSums, d_odata, s);
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				exit(1);
			}
			s /= (NTHREADS * 2);
		}
		cudaStatus = hipMemcpy(&res, d_odata, sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			cudaStatus = hipGetLastError();
			fprintf(stderr, "hipMemcpy getOutput failed! %s\n", hipGetErrorString(cudaStatus));
			exit(1);
		}
	}
	clock_gettime(CLOCK_REALTIME, &time_end);
	double costTime = (time_end.tv_sec - time_start.tv_sec) * 1000 * 1000 * 1000 + time_end.tv_nsec - time_start.tv_nsec;
	printf("duration:%.7lfdms\n", costTime / 1000 / 1000);
	hipFree(d_intermediateSums);
	return res;
}

__host__ int main()
{
	int *h_idata;
	h_idata = (int *)malloc(BYTES);
	for (int i = 0; i < FOURMB; i++)
	{
		h_idata[i] = rand() & 0xff;
	}
	int cpu_result = 0;
	for (int i = 0; i < FOURMB; i++)
	{
		cpu_result += h_idata[i];
	}
	int gpu_result = reduce_start(h_idata);
	printf("cpu_result: %d\n", cpu_result);
	printf("gpu_result: %d\n", gpu_result);
}